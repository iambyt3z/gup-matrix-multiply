
#include <hip/hip_runtime.h>
#include <stdio.h>

#define TILE_SIZE 16

__global__ void
mysgemm(int m, int n, int k, const float *A, const float *B, float* C)
{
    // Allocate shared memory for sub-matrices of A and B
    __shared__ float A_shared[TILE_SIZE][TILE_SIZE];
    __shared__ float B_shared[TILE_SIZE][TILE_SIZE];

    // Calculate the row and column index of the element C
    int row = blockIdx.y * TILE_SIZE + threadIdx.y;
    int col = blockIdx.x * TILE_SIZE + threadIdx.x;
    
    float C_value = 0;

    // Loop over tiles of A and B
    for (int tile = 0; tile < (k + TILE_SIZE - 1) / TILE_SIZE; tile++) {
        // Load tiles into shared memory
        A_shared[threadIdx.y][threadIdx.x] = 
            (float) (row < m && tile * TILE_SIZE + threadIdx.x < k) *
            A[row * k + tile * TILE_SIZE + threadIdx.x];
        
        B_shared[threadIdx.y][threadIdx.x] = 
            (float) (col < n && tile * TILE_SIZE + threadIdx.y < k) *
            B[(tile * TILE_SIZE + threadIdx.y) * n + col];

        __syncthreads();

        // Multiply the two matrices
        for (int kk = 0; kk < TILE_SIZE; kk++) {
            C_value += A_shared[threadIdx.y][kk] * B_shared[kk][threadIdx.x];
        }

        __syncthreads();
    }

    // Write result to C
    if (row < m && col < n) {
        C[row * n + col] = C_value;
    }
}

void 
basicSgemm(int m, int n, int k, const float *A, const float *B, float *C) 
{
    const unsigned int BLOCK_SIZE = TILE_SIZE;

    dim3 dim_block(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dim_grid((n + BLOCK_SIZE - 1) / BLOCK_SIZE, (m + BLOCK_SIZE - 1) / BLOCK_SIZE);

    // Launch the tiled matrix multiplication kernel
    mysgemm<<<dim_grid, dim_block>>>(m, n, k, A, B, C);
}
